#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256
#define BLOCK_SIZE 16

//@@ insert code here
__global__ void histEq(float *input, int height, int width){
  //initialize shared data
  __shared__ blockHistogram[HISTOGRAM_LENGTH];
  __shared__ blockGrayImg[BLOCK_SIZE*BLOCK_SIZE];

  //Set index
  int idx = 

}

int main(int argc, char **argv) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  const char *inputImageFile;

  //@@ Insert more code here

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  wbTime_start(Generic, "Importing data and creating memory on host");
  inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
  hostInputImageData = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);
  wbTime_stop(Generic, "Importing data and creating memory on host");

  //@@ insert code here

  wbSolution(args, outputImage);

  //@@ insert code here

  return 0;
}
